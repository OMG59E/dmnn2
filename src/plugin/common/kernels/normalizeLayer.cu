#include "hip/hip_runtime.h"
/*
 * @Author: xingwg
 * @Date: 2024-10-12 09:59:37
 * @LastEditTime: 2024-10-12 11:59:29
 * @FilePath: /dmnn2/src/plugin/common/kernels/normalizeLayer.cu
 * @Description: 
 * 
 * Copyright (c) 2024 by Chinasvt, All Rights Reserved. 
 */

#include "kernel.h"
#include "../bboxUtils.h"

size_t normalizePluginWorkspaceSize(bool acrossSpatial, int C, int H, int W) {
    if (acrossSpatial)
        return sizeof(float) * C * H * W;
    else
        return (size_t) 0;
}

template<unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta)
__global__ void normalizeNotAcrossSpatialKernel(
        const bool channelShared,
        const int N,
        const int C,
        const int H,
        const int W,
        const float eps,
        const float *scale,
        float *inputData,
        float *outputData) {
    const int dim = C * H * W;
    const int spatialDim = H * W;
    const int tile = 32;
    const int numTile = (spatialDim + tile - 1) / tile;
    for (int n = blockIdx.x; n < N * numTile; n += gridDim.x) {
        float *input = inputData + (n / numTile) * dim;
        float *output = outputData + (n / numTile) * dim;
        __shared__ float sum[tile];
        float localsum = 0.0F;
        for (int i = threadIdx.x; i < tile; i += nthds_per_cta) {
            sum[i] = 0.0F;
        }
        __syncthreads();
        for (int i = threadIdx.x; i < C * tile; i += nthds_per_cta) {
            int row = i / tile;
            int col = (n % numTile) * tile + i % tile;
            float data = 0.0F;
            if (col < spatialDim)
                data = input[row * spatialDim + col];
            localsum += data * data;
        }
        atomicAdd(&sum[threadIdx.x & 31], localsum);
        __syncthreads();
        for (int i = threadIdx.x; i < C * tile; i += nthds_per_cta) {
            int row = i / tile;
            int col = (n % numTile) * tile + i % tile;
            if (col < spatialDim) {
                int offset = row * spatialDim + col;
                output[offset] = input[offset] / sqrt(sum[threadIdx.x & 31] + eps);
            }
        }
        if (channelShared) {
            for (int i = threadIdx.x; i < C * tile; i += nthds_per_cta) {
                int row = i / tile;
                int col = (n % numTile) * tile + i % tile;
                if (col < spatialDim)
                    output[row * spatialDim + col] *= scale[0];
            }
        } else {
            for (int i = threadIdx.x; i < C * tile; i += nthds_per_cta) {
                int row = i / tile;
                int col = (n % numTile) * tile + i % tile;
                if (col < spatialDim)
                    output[row * spatialDim + col] *= scale[row];
            }
        }
    }
}

pluginStatus_t normalizeNotAcrossSpatialGpu(
        hipStream_t stream,
        const bool channelShared,
        const int N,
        const int C,
        const int H,
        const int W,
        const float eps,
        const void *scale,
        const void *inputData,
        void *outputData) {
    const int BS = 128;
    const int GS = 256;
    // assumes warp size == 32
    if (BS % 32 != 0)
        LOG_FATAL("BS %d should be a multiple of 32", BS);
    normalizeNotAcrossSpatialKernel < BS ><<<GS, BS, 0, stream>>>(channelShared, N, C, H, W, eps, (const float *) scale, (float *) inputData, (float *) outputData);
    CUDACHECK(hipGetLastError());
    return STATUS_SUCCESS;
}

__global__ void squareKernel(const int n, const float *x, float *y) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
        y[i] = x[i] * x[i];
    }
}

__global__ void scalChannelKernel(
        const int n,
        const int spatialDim,
        const float *inputData,
        const float *scale,
        float *outputData) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
        // scale factors are indepedent across different channels
        // scale[i / spatialDim]: find the right scale factor for specific channels
        outputData[i] = inputData[i] / scale[i / spatialDim];
    }
}

pluginStatus_t normalizeInference(
        hipStream_t stream,
        hipblasHandle_t handle,
        const bool acrossSpatial,
        const bool channelShared,
        const int N,
        const int C,
        const int H,
        const int W,
        const float eps,
        const void *scale,
        const void *inputData,
        void *outputData,
        void *workspace) {
    const int dim = C * H * W;
    // Normalization is conducted for each sample from the batch indepdently
    if (acrossSpatial) {
        float *input = (float *) const_cast<void *>(inputData);
        float *output = (float *) outputData;
        float *buffer = (float *) workspace;
        for (int n = 0; n < N; ++n) {
            // Take the square of each element in the input
            squareKernel << < (dim + 511) / 512, 512, 0, stream >> > (dim, input, buffer);
            float normsqr = 0.0F;
            // Sum up all the squared elements
            CUBLASCHECK(hipblasSasum(handle, dim, buffer, 1, &normsqr));
            // Make a copy of the input to the output
            CUBLASCHECK(hipblasScopy(handle, dim, input, 1, output, 1));
            // Calculate the inverse of the square root of the sum
            // Use eps to prevent being divided by zero
            normsqr = 1 / sqrt(normsqr + eps);
            // Scale all the outputs by normsqr
            CUBLASCHECK(hipblasSscal(handle, dim, &normsqr, output, 1));
            // If channel shared is true, scale all the outputs
            if (channelShared) {
                CUBLASCHECK(hipblasSscal(handle, dim, (float *) scale, output, 1));
            } else { // Use different scale factors for different channels
                // scale the output according to channels
                scalChannelKernel<<<(dim + 511) / 512, 512, 0, stream >>>(dim, H * W, output, (float *) scale, output);
            }
            // Move cursors
            input += dim;
            output += dim;
        }
        return STATUS_SUCCESS;
    } else { // Normalization ignoring the batch
        return normalizeNotAcrossSpatialGpu(stream, channelShared, N, C, H, W, eps, scale, inputData, outputData);
    }
}
